
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__device__ int IP[64];
__device__ int FP[64];
__device__ int E[48];
__device__ int P[32];
__device__ int SBox[8][64];


// Initial Permutation
int host_IP[64] = {
    57, 49, 41, 33, 25, 17,  9,  1,
    59, 51, 43, 35, 27, 19, 11,  3,
    61, 53, 45, 37, 29, 21, 13,  5,
    63, 55, 47, 39, 31, 23, 15,  7,
    56, 48, 40, 32, 24, 16,  8,  0,
    58, 50, 42, 34, 26, 18, 10,  2,
    60, 52, 44, 36, 28, 20, 12,  4,
    62, 54, 46, 38, 30, 22, 14,  6
};

// Final Permutation
int host_FP[64] = {
    39,  7, 47, 15, 55, 23, 63, 31,
    38,  6, 46, 14, 54, 22, 62, 30,
    37,  5, 45, 13, 53, 21, 61, 29,
    36,  4, 44, 12, 52, 20, 60, 28,
    35,  3, 43, 11, 51, 19, 59, 27,
    34,  2, 42, 10, 50, 18, 58, 26,
    33,  1, 41,  9, 49, 17, 57, 25,
    32,  0, 40,  8, 48, 16, 56, 24
};

// Expansion Function: from 32 bit to 48 bit
int host_E[48] = {
    31,  0,  1,  2,  3,  4,
     3,  4,  5,  6,  7,  8,
     7,  8,  9, 10, 11, 12,
    11, 12, 13, 14, 15, 16,
    15, 16, 17, 18, 19, 20,
    19, 20, 21, 22, 23, 24,
    23, 24, 25, 26, 27, 28,
    27, 28, 29, 30, 31,  0
};

// Permutation
int host_P[32] = {
    15,  6, 19, 20, 28, 11, 27, 16,
     0, 14, 22, 25,  4, 17, 30,  9,
     1,  7, 23, 13, 31, 26,  2,  8,
    18, 12, 29,  5, 21, 10,  3, 24
};

int PC1_LEFT[28] = {
    56, 48, 40, 32, 24, 16,  8,
     0, 57, 49, 41, 33, 25, 17,
     9,  1, 58, 50, 42, 34, 26,
    18, 10,  2, 59, 51, 43, 35,
};

int PC1_RIGHT[28] = {
    62, 54, 46, 38, 30, 22, 14,
     6, 61, 53, 45, 37, 29, 21,
    13,  5, 60, 52, 44, 36, 28,
    20, 12,  4, 27, 19, 11,  3
};

int PC2[48] = {
    13, 16, 10, 23,  0,  4,
     2, 27, 14,  5, 20,  9,
    22, 18, 11,  3, 25,  7,
    15,  6, 26, 19, 12,  1,
    40, 51, 30, 36, 46, 54,
    29, 39, 50, 44, 32, 47,
    43, 48, 38, 55, 33, 52,
    45, 41, 49, 35, 28, 31
};

int Rotations[16] = {1, 1, 2, 2, 2, 2, 2, 2, 1, 2, 2, 2, 2, 2, 2, 1};

// Substitution Boxes
int host_SBox[8][64] = {
    // S1
    {
        14, 4, 13, 1, 2, 15, 11, 8, 3, 10, 6, 12, 5, 9, 0, 7,
        0, 15, 7, 4, 14, 2, 13, 1, 10, 6, 12, 11, 9, 5, 3, 8,
        4, 1, 14, 8, 13, 6, 2, 11, 15, 12, 9, 7, 3, 10, 5, 0,
        15, 12, 8, 2, 4, 9, 1, 7, 5, 11, 3, 14, 10, 0, 6, 13
    },

    // S2
    {
        15, 1, 8, 14, 6, 11, 3, 4, 9, 7, 2, 13, 12, 0, 5, 10,
        3, 13, 4, 7, 15, 2, 8, 14, 12, 0, 1, 10, 6, 9, 11, 5,
        0, 14, 7, 11, 10, 4, 13, 1, 5, 8, 12, 6, 9, 3, 2, 15,
        13, 8, 10, 1, 3, 15, 4, 2, 11, 6, 7, 12, 0, 5, 14, 9
    },

    // S3
    {
        10, 0, 9, 14, 6, 3, 15, 5, 1, 13, 12, 7, 11, 4, 2, 8,
        13, 7, 0, 9, 3, 4, 6, 10, 2, 8, 5, 14, 12, 11, 15, 1,
        13, 6, 4, 9, 8, 15, 3, 0, 11, 1, 2, 12, 5, 10, 14, 7,
        1, 10, 13, 0, 6, 9, 8, 7, 4, 15, 14, 3, 11, 5, 2, 12
    },

    // S4
    {
        7, 13, 14, 3, 0, 6, 9, 10, 1, 2, 8, 5, 11, 12, 4, 15,
        13, 8, 11, 5, 6, 15, 0, 3, 4, 7, 2, 12, 1, 10, 14, 9,
        10, 6, 9, 0, 12, 11, 7, 13, 15, 1, 3, 14, 5, 2, 8, 4,
        3, 15, 0, 6, 10, 1, 13, 8, 9, 4, 5, 11, 12, 7, 2, 14
    },

    // S5
    {
        2, 12, 4, 1, 7, 10, 11, 6, 8, 5, 3, 15, 13, 0, 14, 9,
        14, 11, 2, 12, 4, 7, 13, 1, 5, 0, 15, 10, 3, 9, 8, 6,
        4, 2, 1, 11, 10, 13, 7, 8, 15, 9, 12, 5, 6, 3, 0, 14,
        11, 8, 12, 7, 1, 14, 2, 13, 6, 15, 0, 9, 10, 4, 5, 3
    },

    // S6
    {
        12, 1, 10, 15, 9, 2, 6, 8, 0, 13, 3, 4, 14, 7, 5, 11,
        10, 15, 4, 2, 7, 12, 9, 5, 6, 1, 13, 14, 0, 11, 3, 8,
        9, 14, 15, 5, 2, 8, 12, 3, 7, 0, 4, 10, 1, 13, 11, 6,
        4, 3, 2, 12, 9, 5, 15, 10, 11, 14, 1, 7, 6, 0, 8, 13
    },

    // S7
    {
        4, 11, 2, 14, 15, 0, 8, 13, 3, 12, 9, 7, 5, 10, 6, 1,
        13, 0, 11, 7, 4, 9, 1, 10, 14, 3, 5, 12, 2, 15, 8, 6,
        1, 4, 11, 13, 12, 3, 7, 14, 10, 15, 6, 8, 0, 5, 9, 2,
        6, 11, 13, 8, 1, 4, 10, 7, 9, 5, 0, 15, 14, 2, 3, 12
    },

    // S8
    {
        13, 2, 8, 4, 6, 15, 11, 1, 10, 9, 3, 14, 5, 0, 12, 7,
        1, 15, 13, 8, 10, 3, 7, 4, 12, 5, 6, 11, 0, 14, 9, 2,
        7, 11, 4, 1, 9, 12, 14, 2, 0, 6, 10, 13, 15, 3, 5, 8,
        2, 1, 14, 7, 4, 10, 8, 13, 15, 12, 9, 0, 3, 5, 6, 11
    }
};

__device__
__host__
long long int permutation(long long int data, int data_size, int *table, int table_size) {
    long long int result = 0;
    int i = 0;
    for (; i < table_size; i++) {
        result = (result << 1) + ((data >> (data_size - 1 - table[i])) & 0x1);
    }
    return result;
}

long long int *generate_sub_keys(long long int key, int decrypt) {
    int n_keys = 16;
    long long int *sub_keys = (long long int *) malloc(sizeof(long long int) * n_keys);
    int half_key_length = 28;
    long long int left = permutation(key, 64, PC1_LEFT, half_key_length);
    long long int right = permutation(key, 64, PC1_RIGHT, half_key_length);
    int i = 0;
    for (; i < n_keys; i++) {
        int rotation = Rotations[i];
        left = (((left << rotation) | (left >> (half_key_length - rotation))) & 0xFFFFFFF);
        right = (((right << rotation) | (right >> (half_key_length - rotation))) & 0xFFFFFFF);
        long long int new_key = (left << half_key_length) | right;
        int sub_key_index = (decrypt ? 15 - i : i);
        sub_keys[sub_key_index] = permutation(new_key, half_key_length * 2, PC2, 48);
    }
    return sub_keys;
}

__device__
long long int substitution(long long int data) {
    // data: 48 bit
    long long int result = 0;
    int i = 0;
    for (; i < 8; i++) {
        unsigned int box = data >> (6 * (7 - i)) & 0x3F;
        int outer = ((box & 0x20) >> 4) | (box & 0x1);
        int inner = (box & 0x1E) >> 1;
        result = (result << 4) + SBox[i][(outer << 4) + inner];
    }
    return result;
}

__device__
long int F(unsigned int c, long long int key) {
    long long int lc = c;
    long long int new_c = permutation(lc, 32, E, 48);
    long long int mixed_data = new_c ^ key;
    long long int s_box_result = substitution(mixed_data);
    return permutation(s_box_result, 32, P, 32);
}

__device__
void DES(int index, long long int *MD, long long int *keys) {
    long long int data = permutation(MD[index], 64, IP, 64);
    unsigned int left = data >> 32;
    unsigned int right = (int) data;
    int i = 0;
    for (; i < 16; i++) {
        unsigned int buf = left ^ F(right, keys[i]);
        left = right;
        right = buf;
    }
    data = right;
    data = (data << 32) + left;
    MD[index] = permutation(data, 64, FP, 64);
}

__global__
void kernel_DES(unsigned int quota, unsigned int n_blocks, long long int *MD, long long int *keys) {
    int start_index = (blockIdx.x * blockDim.x + threadIdx.x) * quota;
    int end_index = start_index + quota;
    int i;
    for (i = start_index; i < end_index; i++) {
        if (i >= n_blocks) {
        return;
        }
        DES(i, MD, keys);
    }
}

void runDESCuda(unsigned int n_blocks, long long int *host_MD, long long int *host_sub_keys, int n_cuda_blocks, int n_cuda_threads) {
    unsigned int max_n_threads = 512;
    if (n_cuda_blocks == -1 && n_cuda_threads == -1) {
        n_cuda_threads = max_n_threads;
        n_cuda_blocks = (n_blocks + max_n_threads - 1) / max_n_threads;
    }
    if (n_cuda_threads > max_n_threads) {
        printf("Maximum value of the number of threads is 512. You entered : %d\n", n_cuda_threads);
        return;
    }
    hipMemcpyToSymbol(HIP_SYMBOL(IP), host_IP, sizeof(host_IP));
    hipMemcpyToSymbol(HIP_SYMBOL(FP), host_FP, sizeof(host_FP));
    hipMemcpyToSymbol(HIP_SYMBOL(E), host_E, sizeof(host_E));
    hipMemcpyToSymbol(HIP_SYMBOL(P), host_P, sizeof(host_P));
    hipMemcpyToSymbol(HIP_SYMBOL(SBox), host_SBox, sizeof(host_SBox));

    long long int *MD, *sub_keys;
    hipMalloc((void **) &MD, sizeof(long long int) * n_blocks);
    hipMemcpy(MD, host_MD, sizeof(long long int) * n_blocks, hipMemcpyHostToDevice);
    hipMalloc((void **) &sub_keys, sizeof(long long int) * 16);
    hipMemcpy(sub_keys, host_sub_keys, sizeof(long long int) * 16, hipMemcpyHostToDevice);
    unsigned int quota = n_blocks / (n_cuda_blocks * n_cuda_threads) + 1;
    printf("%u bytes per threads.\n", quota * 64);
    kernel_DES<<<n_cuda_blocks, n_cuda_threads>>>(quota, n_blocks, MD, sub_keys);
    hipMemcpy(host_MD, MD, sizeof(long long int) * n_blocks, hipMemcpyDeviceToHost);

    hipFree(IP);
    hipFree(FP);
    hipFree(E);
    hipFree(P);
    hipFree(SBox);
    hipFree(MD);
    hipFree(sub_keys);
}

unsigned int n_blocks = 0;
void des_with_file(int decrypt, char *in, char *out, char *key, int n_cuda_blocks, int n_cuda_threads) {
    int buf_size = 8 * n_blocks;
    char *buf = (char *) malloc(sizeof(char) * buf_size);
    FILE *in_fp = fopen(in, "rb");
    if (in_fp == NULL) {
        printf("Can't open the in file :%s\n", in);
        return;
    }
    fread(buf, buf_size, 1, in_fp);
    fclose(in_fp);

    long long int *MD = (long long int *) malloc(sizeof(long long int) * n_blocks);
    int i = 0;
    int j = 0;
    for (i = 0; i < n_blocks; i++) {
        long long int block = 0;
        for (j = 0; j < 8; j++) {
            block = (block << 8) + (buf[(i * 8) + j] & 0xFF);
        }
        MD[i] = block;
    }
    long long int binary_key = 0;
    for (i = 0; i < 8; i++) {
        binary_key = (binary_key << 8) + (key[i] & 0xFF);
    }

    long long int *sub_keys = generate_sub_keys(binary_key, decrypt);
    clock_t begin = clock();
    runDESCuda(n_blocks, MD, sub_keys, n_cuda_blocks, n_cuda_threads);
    clock_t end = clock();
    double time_spent = (double) (end - begin) / CLOCKS_PER_SEC;
    printf("CUDA time: %f\n", time_spent);
    free(sub_keys);

    for (i = 0; i < n_blocks; i++) {
        for (j = 0; j < 8; j++) {
            buf[(i * 8) + (7 - j)] = ((MD[i] >> (j * 8)) & 0xFF);
        }
    }
    FILE *out_fp = fopen(out, "wb");
    if (out_fp == NULL) {
        printf("Can't open the out file :%s\n", out);
        return;
    }
    fwrite(buf, buf_size, 1, out_fp);
    fclose(out_fp);
    free(buf);
    free(MD);
}

void encryption(char *in, char *out, char *key, int n_cuda_blocks, int n_cuda_threads) {
    des_with_file(0, in, out, key, n_cuda_blocks, n_cuda_threads);
}

void decryption(char *in, char *out, char *key, int n_cuda_blocks, int n_cuda_threads) {
    des_with_file(1, in, out, key, n_cuda_blocks, n_cuda_threads);
}

int main(int argc, char** argv) {
    if (argc < 6) {
        printf("usage) ./des.out [e|d] <input_file> <output_file> <n_des_block_size> <n_cuda_blocks> <n_cuda_threads>\n");
        printf("example) ./des.out e in.txt out.txt 1\n");
        return -1;
    }
    int n_cuda_blocks;
    int n_cuda_threads;
    sscanf(argv[5], "%d", &n_blocks);
    sscanf(argv[6], "%d", &n_cuda_blocks);
    sscanf(argv[7], "%d", &n_cuda_threads);
    switch(argv[1][0]) {
        case 'e':
            printf("encryption\n");
            encryption(argv[2], argv[3], argv[4], n_cuda_blocks, n_cuda_threads);
            break;
        case 'd':
            printf("decryption\n");
            decryption(argv[2], argv[3], argv[4], n_cuda_blocks, n_cuda_threads);
            break;
        default:
            printf("mode must be 'e' or 'd'\n");
    }
}
